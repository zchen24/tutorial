#include "hip/hip_runtime.h"
// From CUDA for Engineering
// norm/kernel.cu
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include <math.h>
#include <stdio.h>

#define N (128*128)

int main(int argc, char** argv)
{
    thrust::device_vector<float> dvec_x(N, 1.0f);
    float norm = sqrt(thrust::inner_product(dvec_x.begin(), dvec_x.end(), dvec_x.begin(), 0.0f));
    printf("norm = %.0f\n", norm);
    return 0;
}
