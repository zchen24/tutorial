// CUDA by Example
// Ch10: page-locked (pinned) host memory

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

static void HandleError(hipError_t err,
    const char *file,
    int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
            file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define SIZE (10*1024*1024)

float cuda_malloc_test(int size, bool up)
{
    hipEvent_t start, stop;
    int *a, *d_a;
    float d_t;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    a = (int*)malloc(size*sizeof(*a));
    hipMalloc(&d_a, size * sizeof(*a));

    hipEventRecord(start, 0);
    for (int i = 0; i < 100; i++) {
        if (up) {
            hipMemcpy(d_a, a, size * sizeof(*a), hipMemcpyHostToDevice);
        }
        else {
            hipMemcpy(a, d_a, size * sizeof(*a), hipMemcpyDeviceToHost);
        }
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&d_t, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(a);
    hipFree(d_a);

    return d_t;
}

float cuda_host_alloc_test(int size, bool up)
{
    hipEvent_t start, stop;
    int *a, *d_a;
    float d_t;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipHostAlloc(&a, size*sizeof(*a), hipHostMallocDefault);
    hipMalloc(&d_a, size*sizeof(*a));

    hipEventRecord(start, 0);
    for (int i = 0; i < 100; i++) {
        if (up) {
            hipMemcpy(d_a, a, size * sizeof(*a), hipMemcpyHostToDevice);
        }
        else {
            hipMemcpy(a, d_a, size * sizeof(*a), hipMemcpyDeviceToHost);
        }
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&d_t, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipHostFree(a);
    hipFree(d_a);
    return d_t;
}



int main()
{
    float elapsedTime;
    float MB = (float)100 * SIZE * sizeof(int) / 1024 / 1024;
    
    elapsedTime = cuda_malloc_test(SIZE, true);
    printf("Time using cudaMalloc: %3.1f ms\n", elapsedTime);
    printf("\t MB/s during copy up: %3.1f\n", MB/(elapsedTime/1000));

    elapsedTime = cuda_malloc_test(SIZE, false);
    printf("Time using cudaMalloc: %3.1f ms\n", elapsedTime);
    printf("\t MB/s during copy down: %3.1f\n", MB / (elapsedTime / 1000));

    elapsedTime = cuda_host_alloc_test(SIZE, true);
    printf("Time using cudaHostAlloc: %3.1f ms\n", elapsedTime);
    printf("\t MB/s during copy up: %3.1f\n", MB / (elapsedTime / 1000));

    elapsedTime = cuda_host_alloc_test(SIZE, false);
    printf("Time using cudaHostAlloc: %3.1f ms\n", elapsedTime);
    printf("\t MB/s during copy down: %3.1f\n", MB / (elapsedTime / 1000));


    return 0;
}
