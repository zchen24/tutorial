#include "hip/hip_runtime.h"
// CUDA by Example
// Ch10.4: using a single CUDA stream

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>

#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)


static void HandleError(hipError_t err,
    const char *file,
    int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
            file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__ void kernel(int* d_a, int* d_b, int* d_c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float as = (d_a[idx] + d_a[idx1] + d_a[idx2]) / 3.0f;
        float bs = (d_b[idx] + d_b[idx1] + d_b[idx2]) / 3.0f;
        d_c[idx] = (as + bs) / 2;
    }
}


int main()
{
    hipDeviceProp_t prop;
    int device_id = 0;
    HANDLE_ERROR(hipGetDevice(&device_id));
    HANDLE_ERROR(hipGetDeviceProperties(&prop, device_id));
    if (!prop.deviceOverlap) {
        printf("Device will not handle overlaps, exiting...");
        return -1;
    }

    hipEvent_t start, stop;
    float elapsedTime;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    hipStream_t stream0, stream1;
    HANDLE_ERROR(hipStreamCreate(&stream0));
    HANDLE_ERROR(hipStreamCreate(&stream1));
    HANDLE_ERROR(hipEventRecord(start));

    int *h_a, *h_b, *h_c;
    int *d_a0, *d_b0, *d_c0;
    int *d_a1, *d_b1, *d_c1;
    hipMalloc(&d_a0, N * sizeof(int));
    hipMalloc(&d_b0, N * sizeof(int));
    hipMalloc(&d_c0, N * sizeof(int));
    hipMalloc(&d_a1, N * sizeof(int));
    hipMalloc(&d_b1, N * sizeof(int));
    hipMalloc(&d_c1, N * sizeof(int));
    hipHostMalloc(&h_a, FULL_DATA_SIZE * sizeof(int));
    hipHostMalloc(&h_b, FULL_DATA_SIZE * sizeof(int));
    hipHostMalloc(&h_c, FULL_DATA_SIZE * sizeof(int));

    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        h_a[i] = rand();
        h_b[i] = rand();
    }

    // loop over full data in chunks
    for (int i = 0; i < FULL_DATA_SIZE; i += 2*N) {
#if 1
        // WRONG!
        HANDLE_ERROR(hipMemcpyAsync(d_a0, h_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0));
        HANDLE_ERROR(hipMemcpyAsync(d_b0, h_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0));
        kernel << <N / 256, 256, 0, stream0 >> > (d_a0, d_b0, d_c0);
        HANDLE_ERROR(hipMemcpyAsync(h_c + i, d_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0));

        HANDLE_ERROR(hipMemcpyAsync(d_a1, h_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1));
        HANDLE_ERROR(hipMemcpyAsync(d_b1, h_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1));        
        kernel << <N / 256, 256, 0, stream1 >> > (d_a1, d_b1, d_c1);        
        HANDLE_ERROR(hipMemcpyAsync(h_c + i + N, d_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1));
#else
        HANDLE_ERROR(hipMemcpyAsync(d_a0, h_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0));        
        HANDLE_ERROR(hipMemcpyAsync(d_b0, h_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0));
        HANDLE_ERROR(hipMemcpyAsync(d_a1, h_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1));
        HANDLE_ERROR(hipMemcpyAsync(d_b1, h_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1));                
        kernel << <N / 256, 256, 0, stream0 >> > (d_a0, d_b0, d_c0);       
        kernel << <N / 256, 256, 0, stream1 >> > (d_a1, d_b1, d_c1);
        HANDLE_ERROR(hipMemcpyAsync(h_c + i, d_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0));
        HANDLE_ERROR(hipMemcpyAsync(h_c + i + N, d_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1));
#endif
    }
    HANDLE_ERROR(hipStreamSynchronize(stream0));
    HANDLE_ERROR(hipStreamSynchronize(stream1));

    HANDLE_ERROR(hipEventRecord(stop));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Time taken: %3.1f ms\n", elapsedTime);

    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));
    HANDLE_ERROR(hipStreamDestroy(stream0));
    HANDLE_ERROR(hipStreamDestroy(stream1));
    HANDLE_ERROR(hipFree(d_a0));
    HANDLE_ERROR(hipFree(d_b0));
    HANDLE_ERROR(hipFree(d_c0));
    HANDLE_ERROR(hipFree(d_a1));
    HANDLE_ERROR(hipFree(d_b1));
    HANDLE_ERROR(hipFree(d_c1));
    HANDLE_ERROR(hipHostFree(h_a));
    HANDLE_ERROR(hipHostFree(h_b));
    HANDLE_ERROR(hipHostFree(h_c));

    return 0;
}
