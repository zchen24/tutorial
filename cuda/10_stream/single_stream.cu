#include "hip/hip_runtime.h"
// CUDA by Example
// Ch10.4: using a single CUDA stream

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>

#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)


static void HandleError(hipError_t err,
    const char *file,
    int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
            file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__ void kernel(int* d_a, int* d_b, int* d_c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float as = (d_a[idx] + d_a[idx1] + d_a[idx2]) / 3.0f;
        float bs = (d_b[idx] + d_b[idx1] + d_b[idx2]) / 3.0f;
        d_c[idx] = (as + bs) / 2;
    }
}


int main()
{
    hipDeviceProp_t prop;
    int device_id = 0;
    HANDLE_ERROR(hipGetDevice(&device_id));
    HANDLE_ERROR(hipGetDeviceProperties(&prop, device_id));
    if (!prop.deviceOverlap) {
        printf("Device will not handle overlaps, exiting...");
        return -1;
    }

    hipEvent_t start, stop;
    float elapsedTime;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    hipStream_t stream;
    HANDLE_ERROR(hipStreamCreate(&stream));
    HANDLE_ERROR(hipEventRecord(start));

    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_c, N * sizeof(int));
    hipHostMalloc(&h_a, FULL_DATA_SIZE * sizeof(int));
    hipHostMalloc(&h_b, FULL_DATA_SIZE * sizeof(int));
    hipHostMalloc(&h_c, FULL_DATA_SIZE * sizeof(int));

    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        h_a[i] = rand();
        h_b[i] = rand();
    }

    // loop over full data in chunks
    for (int i = 0; i < FULL_DATA_SIZE; i += N) {
        HANDLE_ERROR(hipMemcpyAsync(d_a, h_a+i, N*sizeof(int), hipMemcpyHostToDevice, stream));
        HANDLE_ERROR(hipMemcpyAsync(d_b, h_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream));
        kernel <<<N/256, 256, 0, stream>>> (d_a, d_b, d_c);
        HANDLE_ERROR(hipMemcpyAsync(h_c+i, d_c, N * sizeof(int), hipMemcpyDeviceToHost, stream));
    }
    HANDLE_ERROR(hipStreamSynchronize(stream));

    HANDLE_ERROR(hipEventRecord(stop));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Time taken: %3.1f ms\n", elapsedTime);

    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));
    HANDLE_ERROR(hipStreamDestroy(stream));
    HANDLE_ERROR(hipFree(d_a));
    HANDLE_ERROR(hipFree(d_b));
    HANDLE_ERROR(hipFree(d_c));
    HANDLE_ERROR(hipHostFree(h_a));
    HANDLE_ERROR(hipHostFree(h_b));
    HANDLE_ERROR(hipHostFree(h_c));

    return 0;
}
