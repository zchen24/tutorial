// From CUDA for Engineers
// Listing 4.3

#include <hip/hip_runtime.h>
#include <iostream>


#define W   500
#define H   500
#define TX  32      // thread per block along x
#define TY  32      // thread per block along y


__global__
void distanceKernel(float *d_out, int w, int h, float2 pos)
{
	const int c = blockIdx.x * blockDim.x + threadIdx.x;
	const int r = blockIdx.y * blockDim.y + threadIdx.y;
	const int i = r * w + c;
	if ( (c >= w) || (r >= h) ) return;

	// compute the distance
	d_out[i] = sqrtf((c - pos.x) * (c - pos.x) + (r - pos.y) * (r - pos.y));
}


int main()
{
    float *out = (float*)calloc(W*H, sizeof(float));
    float *d_out = 0;
    hipMalloc(&d_out, W*H*sizeof(float));

    const float2 pos = {0.0f, 0.0f};
    const dim3 blockSize(TX, TY);
    const int bx = (W + TX - 1) / TX;
    const int by = (W + TY - 1) / TY;
    const dim3 gridSize = dim3(bx, by);

    distanceKernel<<<gridSize, blockSize>>>(d_out, W, H, pos);
    hipDeviceSynchronize();
    hipMemcpy(out, d_out, W*H*sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "bx = " << bx << "  by = " << by << "\n";
    std::cout << "out[0] = " << out[0] << "   out[end] = " << out[W*H-1] << '\n';

    free(out);
    hipFree(d_out);

    return 0;
}



