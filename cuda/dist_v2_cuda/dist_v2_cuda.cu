// From CUDA for Engineering
// dist_v2/kernel.cu

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>

#define TPB 32
#define N 256000
#define M 5   // number of times to do cudaMemcpy

#define DEBUG 0

__device__ 
float distance(float x1, float x2)
{
    return sqrt((x2 - x1) * (x2 - x1));
}

__global__
void distanceKernel(float *d_out, float *d_in, float ref, int len)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= len) { return; }
    const float x = d_in[i];
    d_out[i] = distance(x, ref);
}

void distanceArray(float *out, float *in, float ref, int len)
{
    // alloc cuda memory
    float *d_in = 0;
    float *d_out = 0;
    hipMalloc(&d_in, len * sizeof(float));
    hipMalloc(&d_out, len * sizeof(float));

    // memcpy to device
    struct timespec t0 = {0,0};
    struct timespec t1 = {0,0};
    clock_gettime(CLOCK_REALTIME, &t0);
    for (int i = 0; i < M; i++) {
        hipMemcpy(d_in, in, len * sizeof(float), hipMemcpyHostToDevice);
    }
    clock_gettime(CLOCK_REALTIME, &t1);
    std::cout << "Data transfer time (ms) = " << (t1.tv_sec-t0.tv_sec)*1e3 + (t1.tv_nsec-t0.tv_nsec)/1e6  << "\n";


    // call wrapper
    clock_gettime(CLOCK_REALTIME, &t0);
    distanceKernel<<<(len+TPB-1)/TPB, TPB>>>(d_out, d_in, ref, len);
    hipDeviceSynchronize();
    clock_gettime(CLOCK_REALTIME, &t1);
    std::cout << "Kernel time (ms) = " << (t1.tv_sec-t0.tv_sec)*1e3 + (t1.tv_nsec-t0.tv_nsec)/1e6  << "\n";

    // memcpy from device
    hipMemcpy(out, d_out, len * sizeof(float), hipMemcpyDeviceToHost);

    // free cuda memory
    hipFree(d_in);
    hipFree(d_out);
}

float scale(int i, int n) {
    return ((float)i) / (n - 1);
}

int main()
{
    std::cout << "dist_v2_cuda\n";

    const float ref = 0.5f;
    float *in = (float*)calloc(N, sizeof(float));
    float *out = (float*)calloc(N, sizeof(float));

    for (int i = 0; i < N; i++) {
        in[i] = scale(i, N);
    }
    distanceArray(out, in, ref, N);

#if DEBUG
    std::cout << std::fixed << std::setprecision(4);
    for (int i = 0; i < N; i++) {
        std::cout << "i = " << i << "\tin: " << in[i] << "\tout: " << out[i] << "\n";
    }
#endif

    free(in);
    free(out);
    return 0;
}