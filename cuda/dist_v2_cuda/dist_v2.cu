#include "hip/hip_runtime.h"
// From CUDA for Engineering
// dist_v2/kernel.cu

#include "dist_v2.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#define TPB 32

__device__ 
float distance(float x1, float x2)
{
    return sqrt((x2 - x1) * (x2 - x1));
}

__global__
void distanceKernel(float *d_out, float *d_in, float ref)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const float x = d_in[i];
    d_out[i] = distance(x, ref);
}

void distanceArray(float *out, float *in, float ref, int len)
{
    // alloc cuda memory
    float *d_in = 0;
    float *d_out = 0;
    hipMalloc(&d_in, len * sizeof(float));
    hipMalloc(&d_out, len * sizeof(float));

    // memcpy to device
    hipMemcpy(d_in, in, len * sizeof(float), hipMemcpyHostToDevice);

    // call wrapper
    distanceKernel<<<len/TPB, TPB>>>(d_out, d_in, ref);

    // memcpy from device
    hipMemcpy(out, d_out, len * sizeof(float), hipMemcpyDeviceToHost);

    // free cuda memory
    hipFree(d_in);
    hipFree(d_out);
}