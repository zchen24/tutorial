// From CUDA for Engineer
// Listing D.5

#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

int main()
{
    int numDevices = 0;
    hipGetDeviceCount(&numDevices);
    cout << "Number of devices = " << numDevices << "\n";
    for (int i = 0; i < numDevices; i++) {
        struct hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        cout << "Device Number: " << i << "\n"
             << "Device name: " << prop.name << "\n"
             << "Compute capability: " << prop.major << "." << prop.minor << "\n"
             << "MultiProcessor count: " << prop.multiProcessorCount << "\n"
             << "Maximum threads/block: " << prop.maxThreadsPerBlock << "\n"
             << "Shared memory/block: " << prop.sharedMemPerBlock/1024.0 << " KBytes\n"
             << "Total global memory: " << prop.totalGlobalMem/1e9 << " Gbs\n"
             << "Total constant memory: " << prop.totalConstMem/1024.0 << " KBytes\n";        
    }

    return 0;
}