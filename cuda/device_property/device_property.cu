// From CUDA for Engineer
// Listing D.5

#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

int main()
{
    int numDevices = 0;
    hipGetDeviceCount(&numDevices);
    cout << "Number of devices = " << numDevices << "\n";
    for (int i = 0; i < numDevices; i++) {
        struct hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        cout << "Device Number: " << i << "\n"
             << "Device name: " << prop.name << "\n"
             << "Compute capability: " << prop.major << "." << prop.minor << "\n"
             << "Maximum threads/block: " << prop.maxThreadsPerBlock << "\n"
             << "Shared memory/block: " << prop.sharedMemPerBlock << " bytes\n"
             << "Total global memory: " << prop.totalGlobalMem/1e9 << " Gbs\n";
    }

    return 0;
}