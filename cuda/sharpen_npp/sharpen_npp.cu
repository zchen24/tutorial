#define cimg_display 0
#include <iostream>
#include <hip/hip_runtime.h>
#include <npp.h>
#include "../CImg.h"

#define kNumCh 3


void sharpenNPP(Npp8u *arr, int w, int h) {
    Npp8u *d_in = 0;
    Npp8u *d_out = 0;
    Npp32f *d_filter = 0;
    const Npp32f filter[9] = {-1.0, -1.0, -1.0,
            -1.0, 9.0, -1.0,
            -1.0, -1.0, -1.0};
    hipMalloc(&d_out, kNumCh*w*h*sizeof(Npp8u));
    hipMalloc(&d_in, kNumCh*w*h*sizeof(Npp8u));
    hipMalloc(&d_filter, 9*sizeof(Npp32f));

    hipMemcpy(d_in, arr, kNumCh*w*h*sizeof(Npp8u), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, filter, 9*sizeof(Npp32f), hipMemcpyHostToDevice);

    const NppiSize oKernelSize = {3, 3};
    const NppiPoint oAnchor = {1, 1};
    const NppiSize oSrcSize = {w, h};
    const NppiPoint oSrcOffset = {0, 0};
    const NppiSize oSizeROI = {w, h};

    nppiFilterBorder32f_8u_C3R(d_in, kNumCh*w*sizeof(Npp8u), oSrcSize,
            oSrcOffset, d_out, kNumCh*w*sizeof(Npp8u),
            oSizeROI, d_filter, oKernelSize, oAnchor, NPP_BORDER_REPLICATE);

    // size
    hipMemcpy(arr, d_out, kNumCh*w*h*sizeof(Npp8u), hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_filter);
}


int main()
{
    std::cout << "sharpen_npp\n";
    int device_count = 0;
    hipGetDeviceCount(&device_count);
    if (device_count < 1) {
        std::cerr << "Failed to find CUDA device. Exiting ... \n";
        return -1;
    }

    cimg_library::CImg<unsigned char> image("butterfly.bmp");
    const int w = image.width();
    const int h = image.height();
    std::cout << "image w = " << w << "  h = " << h << "\n";
    Npp8u *arr = (Npp8u*)malloc(kNumCh*w*h*sizeof(Npp8u));

    // copy into arr
    for (int r = 0; r < h; ++r) {
        for (int c = 0; c < w; c++) {
            for (int ch = 0; ch < kNumCh; ch++) {
                arr[kNumCh*(r*w+c) + ch] = image(c, r, ch);
            }
        }
    }

    sharpenNPP(arr, w, h);

    for (int r = 0; r < h; r++) {
        for (int c = 0; c < w; c++) {
            for (int ch = 0; ch < kNumCh; ch++) {
                image(c, r, ch) = arr[kNumCh*(r*w+c) + ch];
            }
        }
    }

    image.save_bmp("out.bmp");
    free(arr);
    return 0;
}
